#include <iostream>
#include <stddef.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

// Macro to catch CUDA errors
#define CUDA_SAFE_CALL( call) do {                                             \
    hipError_t err = call;                                                      \
    if (hipSuccess != err) {                                                  \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",          \
                __FILE__, __LINE__, hipGetErrorString( err) );                \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

// Macro to catch cufft errors
#define CUFFT_SAFE_CALL( call) do {                                            \
    hipfftResult err = call;                                                    \
    if (err != HIPFFT_SUCCESS) {                                                \
        fprintf(stderr, "Cufft error in file '%s' in line %i : %s.\n",         \
                __FILE__, __LINE__, "error" );                                 \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

/*
   typedef enum cufftResult_t {
   CUFFT_SUCCESS        = 0x0,
   CUFFT_INVALID_PLAN   = 0x1,
   CUFFT_ALLOC_FAILED   = 0x2,
   CUFFT_INVALID_TYPE   = 0x3,
   CUFFT_INVALID_VALUE  = 0x4,
   CUFFT_INTERNAL_ERROR = 0x5,
   CUFFT_EXEC_FAILED    = 0x6,
   CUFFT_SETUP_FAILED   = 0x7,
   CUFFT_INVALID_SIZE   = 0x8,
   CUFFT_UNALIGNED_DATA = 0x9
   } cufftResult;
 */

using namespace std;

int main(int argc, char* argv[]) 
{
    if (!(argc == 2 || argc == 3)) { 
        cerr << "usage: ./fft N D, where N*1024 is number of elems, "
            << "and D is device number" << endl;
        exit(-1);
    }
    if (argc == 3) {
        hipSetDevice(atoi(argv[2]));
    }

    int count = atoi(argv[1]) * 1024;
    size_t bytes = count * sizeof(hipfftDoubleComplex);

    // Allocate host memory for the signal
    hipfftDoubleComplex* h_signal = new hipfftDoubleComplex[count];

    // Initalize the memory for the signal
    for (unsigned int i = 0; i < count; i++) {
        h_signal[i].x = 1.;
        h_signal[i].y = 0.;
    }

    // Allocate device memory for signal
    hipfftDoubleComplex* d_signal;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_signal, bytes));
    CUDA_SAFE_CALL(hipMemcpy(d_signal, h_signal, bytes,
                hipMemcpyHostToDevice));

    // CUFFT plan
    hipfftHandle plan;
    CUFFT_SAFE_CALL(hipfftPlan1d(&plan, count, HIPFFT_Z2Z, 1));

    // Transform signal -- warm up
    CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_signal, d_signal, HIPFFT_FORWARD));

    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    CUDA_SAFE_CALL(hipEventRecord(start, 0));
    for (int i = 0; i < 100; i++) {
        CUFFT_SAFE_CALL(hipfftExecZ2Z(plan, d_signal, d_signal, HIPFFT_FORWARD));
    }
    hipEventRecord(stop, 0);
    CUDA_SAFE_CALL(hipEventSynchronize(stop));

    float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, start, stop);
    elapsedTime *= 1.e-3;
    cout << count << ", " << elapsedTime / 100.0f << endl;

    // Copy device memory to host
    hipfftDoubleComplex* h_convolved_signal = h_signal;
    CUDA_SAFE_CALL(hipMemcpy(h_convolved_signal, d_signal, bytes, 
                hipMemcpyDeviceToHost));


    //Destroy CUFFT context
    CUFFT_SAFE_CALL(hipfftDestroy(plan));

    // cleanup memory
    delete[] h_signal;
    CUDA_SAFE_CALL(hipFree(d_signal));
}
